#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void hello(void) {
    printf("Hello World from GPU!\n");
}

int main() {
    hello<<<1, 10>>>();
    hipDeviceSynchronize();
    return 1;
}